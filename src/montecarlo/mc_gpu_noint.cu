#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <unistd.h>
#include "../param/param.h"
#include "definitions.h"

#include <iostream>
#include <vector>
#include <algorithm>
#include "functions.h"
#include "hip/device_functions.h"
#include "memory.h"

using namespace std;
const int Ntot  = imax*jmax*kmax;
const int N     = Ntot/num_streams/num_gpu/p_col;        //numer of cells calculated per stream per GPU 
const int NS    = jmax*kmax/num_streams/num_gpu/p_col;   //number of surface cells per stream per GPU
myfloat kP[nT] ,prob[nT][nB] ,probg[nT][nB][nQ];

// global declaration of 2D float texture (visible for host and device code)
texture<myfloat, hipTextureType2D, hipReadModeElementType> tex_Ib;
__device__ __constant__ myfloat probd[2][nB];
__device__ __constant__ myfloat probs[2][nB];
__device__ myfloat solution[num_streams][N];
__device__ myfloat variance[num_streams][N];
__device__ myfloat s_surf[num_streams][NS][2];
__device__ myfloat v_surf[num_streams][NS][2];
#if grey==0
__device__ __constant__ myfloat probq[nQ] = {0.09654009,0.19127881,0.28602321,0.37719708,0.46484917,0.5491611,
                0.62635499,0.69870079,0.76452301,0.8232071,0.87420516,0.91704105,0.95131492,0.97670699,0.99298138,
                1.0};
#endif
#if srt==1
void sort_idx(NarrowBand *narrBand);
#endif

// pointer definitions
Gridn *gridGPU[num_gpu];
EmissSpec *Ibw_d[num_gpu];
hipArray *cuArrayI[num_gpu], *cuArrayP[num_gpu];
hipTextureObject_t *tex_tempf_d[num_gpu];
int *idx_d[num_gpu];
myfloat *wvc_d[num_gpu];
myfloat *Tnb_d[num_gpu];
hipTextureObject_t *tex_d[num_gpu];
hipTextureObject_t *tex_prob_d[num_gpu];
hipTextureObject_t *tex_prob_s[num_gpu];


__global__ void kernel_fluid(hiprandDirectionVectors32_t *rngVectors, Gridn *my_grid, myfloat *wvc_d, myfloat *Tnb_t, int n, int ns, int stream, myfloat kappamax, myfloat Tmax, int *idx_nb,
		hipTextureObject_t *tex_Tf, hipTextureObject_t *tex, hipTextureObject_t *tex_prob,
		int gpu, int zstart, EmissSpec *Ibw)
{

	Count cnt;

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	volatile int tx = threadIdx.x;

	// for the moment ray stays in the register memory, if it spills,
	// we have to move it to the shared memory (it should fit in register memory)
	Beam ray;
	// we start defining the ray-Ibmax that is equal for every position

	__const__ myfloat Tnb[] = {Tnb_t[1] - Tnb_t[0], (Tnb_t[0])/(Tnb_t[1] - Tnb_t[0])};
	myfloat abscotemp1,abscotemp2;
	myfloat ratioI,invTi4;
	myfloat pos;
	myfloat Ibmax = 4.0f * kappamax * pow4(Tmax) * stefan / photmax;


	//define a narrow band
	myfloat wvc;
	__shared__ hiprandState_t state[blockdim];
	// this is basically the loop over the indices in the CPU.
	// the indices are found in relation with the stream-block-Thread design.
	// this is a grid-stride loop, hence every thread will do multiple cells
       	hiprand_init(1234, (tid+1)*(stream+1)*(1+gpu)*zstart, 0, &state[tx]);  // 	Initialize CURAND
#if random==1 
	hiprandStateSobol32_t state1,state2;
        hiprand_init(rngVectors[0],tid,&state1);
        hiprand_init(rngVectors[1],tid,&state2);
#endif
#if srt == 1
	kernel_find(&cnt, Tnb_t, Tmax, &state[tx], tex_prob);
#endif
	for (int idx = tid; idx < n; idx += blockDim.x * gridDim.x)
	{
		solution[stream][idx] = 0;
		ray.i = (idx / (kmax*jmax/num_gpu/p_col) + 1) + (stream)*(imax/ns);
		ray.j = (idx / (kmax/p_col) + 1 - (ray.i-1-(stream)*imax/ns)*(jmax/num_gpu));
		ray.k = (idx - (kmax/p_col) * (ray.j-1 + (ray.i-1-(stream)*imax/ns)*(jmax/num_gpu)) + 1) + zstart-1;
                ray.j+= gpu*(jmax/num_gpu);
               
		ray.Ti = tex3D<myfloat>(*tex_Tf, ray.i+0.5f, ray.j+0.5f, ray.k+0.5f);

		myfloat De_var[nVar];

		for (int v = 0; v<nVar; v++)
		{
			De_var[v] = 0;
			int g=0;
			int nb=0;
			int countnb=0;
			int countg=0;

			myfloat De_OERMc=0;
			/***********  ENTERING THE PHOTON LOOP, DEVIDED INTO SUB-BUNDLES *****************/

			for (int h=0; h<photmax; h++)
			{
				ray.xp = my_grid[0].x[ray.i];
				ray.yp = my_grid[0].y[ray.j];
				ray.zp = my_grid[0].z[ray.k];
				ray.ic = ray.i;
				ray.jc = ray.j;
				ray.kc = ray.k;
				int flag[3];
				
				// Now we have to deal with random numbers, selecting the angles
#if random==1
	                        emiss_angSobol(&ray, &state1, &state2);
#else
				emiss_ang(&ray, &state[tx]);
#endif

				if(ray.sx == 0)
					ray.sx = 1e-10;
				if(ray.sy == 0)
					ray.sy = 1e-10;
				if(ray.sz == 0)
					ray.sz = 1e-10;
				flag[0] = (int)(ray.sx<0);
				flag[1] = (int)(ray.sy<0);
				flag[2] = (int)(ray.sz<0);

				//Find the narrowband index and the g index, based on random or based on the previous count
#if srt == 1
				wave_find(&nb,&g,&countnb,&countg,idx_nb,&cnt);
#else
				// non sorted monte carlo
				int tm =  (int) ((Tmax - Tnb_t[0])/(Tnb_t[1]-Tnb_t[0]));
				find_band(Tnb_t,Tmax,&g,&nb,&state[tx],tm, tex_prob);
#endif
				// the ray emanating from cell i,j,k is completely defined now
				// now we have to march it
				// initializing transmissivity

				ray.tra = 1.0f;
				wvc  =  wvc_d[nb];

				Emission emiss;

				// wall emissions
#if adiabw == 0
                                emiss.west  =  Ibw->west[nb];
#endif
#if adiabe == 0
                                emiss.east  =  Ibw->east[nb];
#endif
#if bdt != 1
				emiss.top   =  Ibw->top[nb];
#endif
#if bdb != 1
				emiss.bot   =  Ibw->bot[nb];
#endif
#if bdn != 1
				emiss.north =  Ibw->north[nb];
#endif
#if bds != 1
				emiss.south =  Ibw->south[nb];
#endif

				// Calculate parameters for calculation of De_OERMc in march_ray
				pos = ((ray.Ti - Tnb_t[0])/(Tnb_t[1]-Tnb_t[0]));
				abscotemp2 = tex3D<myfloat>(*tex, nb+0.5f, pos+0.5f, g+0.5f);
				invTi4 = 1.0f / tex2D(tex_Ib,nb+0.5f ,pos+0.5f);

				pos = ((Tmax - Tnb_t[0])/(Tnb_t[1]-Tnb_t[0]));
				abscotemp1 = tex3D<myfloat>(*tex, nb+0.5f, pos+0.5f, g+0.5f);
				ratioI = (1.0f / invTi4) / tex2D(tex_Ib,nb+0.5f ,pos+0.5f) * abscotemp2 / abscotemp1 ;

				// Adjust the cells according to the scattering of the ray
				// if the ray originates from the negative cell boundary and it
				// is going in the negative direction, shift cells
				// if ( flag[0] && ray.xp == my_grid[0].xu[ray.ic - 1] )
				// 	ray.ic = ray.ic - 1;
				// if ( flag[1] && ray.yp == my_grid[0].yu[ray.jc - 1] )
				// 	ray.jc = ray.jc - 1;
				// if ( flag[2] && ray.zp == my_grid[0].zu[ray.kc - 1] )
				// 	ray.kc = ray.kc - 1;
				// Do the same thing if it is happening in the positive direction
				// if ( !flag[0] && ray.xp == my_grid[0].xu[ray.ic] )
				// 	ray.ic = ray.ic + 1;
				// if ( !flag[1] && ray.yp == my_grid[0].yu[ray.jc] )
				// 	ray.jc = ray.jc + 1;
				// if ( !flag[2] && ray.zp == my_grid[0].zu[ray.kc] )
				// 	ray.kc = ray.kc + 1;

				/**************************************************************************************/
				/************************** ENTERING THE MARCHING LOOP ********************************/
				/**************************************************************************************/

				//loop on all grids with maximum counter my_grid[grd].sm
				for(int grd = 0; grd < grid_num; grd++)
				{
					march_ray(&ray, nb, g, flag, &De_OERMc, Ibmax, my_grid[grd], ratioI, invTi4,
							*tex_Tf, emiss, wvc, Tnb, tex, toll);

					if(grd<grid_num-1) {
						//adapting grid
						ray.ic += my_grid[grd].im/my_grid[grd+1].im-1;
						ray.jc += my_grid[grd].jm/my_grid[grd+1].jm-1;
						ray.kc += my_grid[grd].km/my_grid[grd+1].km-1;
						ray.ic /= my_grid[grd].im/my_grid[grd+1].im;
						ray.jc /= my_grid[grd].jm/my_grid[grd+1].jm;
						ray.kc /= my_grid[grd].km/my_grid[grd+1].km;
					}
				};

			}

			/***********  OUT OF THE PHOTON LOOP ****************************/
			solution[stream][idx] += De_OERMc;
			De_var[v] = De_OERMc;
		}
		/***********  OUT OF THE VARIANCE LOOP ****************************/

		solution[stream][idx] /= nVar;
		variance[stream][idx] = 0;
		for (int v = 0; v<nVar; v++)
		{
			variance[stream][idx] += 1./(nVar-0.5) * 1./(nVar-1) * (De_var[v] - solution[stream][idx]) * (De_var[v] - solution[stream][idx]);
		}
	}
	/***********  OUT OF THE GRID-STRIDE LOOP ****************************/

}

__device__ __forceinline__ void march_ray(Beam *ray, int nb, int g, int *flag, myfloat *De_OERMc,
		myfloat Ibmax, Gridn grid, myfloat ratioI, myfloat invTi4, hipTextureObject_t tex_Tf,
		Emission emiss, myfloat wvc, const myfloat Tnb[], hipTextureObject_t *tex, myfloat tollerance)
{

	int counter = 0;
	myfloat t1,t2,t3,pos,tf;
	while ( (ray->tra > tollerance) && (counter<grid.sm) )
	{

//		if ( flag[0] && ray->xp <= grid.xu[ray->ic - 1] ) {
//			ray->ic = ray->ic - 1;
//			if(ray->ic == 0) {
//				if(bdw==1)	{ray->ic = grid.im; ray->xp = Lx;}
//				else if(bdw==2) {
//#if adiabw == 1
//                                        t2         = (ray->jc-0.5f)*(jmax/grid.jm)+1.0f;
//                                        t3         = (ray->kc-0.5f)*(kmax/grid.km)+1.0f;
//					tf         = tex3D<myfloat>(tex_Tf,1.0f,t2,t3);
//					pos        = (tf/Tnb[0] - Tnb[1]);
//					emiss.west = tex2D(tex_Ib, nb+0.5f ,pos+0.5f);
//#endif
//					*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.west - 1.0f )* ratioI;
//					ray->tra = 0;
//					break; } } }
//		if ( flag[1] && ray->yp <= grid.yu[ray->jc - 1] ) {
//			ray->jc = ray->jc - 1;
//			if(ray->jc == 0) {
//				if(bds==1)	{ray->jc = grid.jm; ray->yp = Ly;}
//				else if(bds==2) {
//					*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.south - 1.0f )* ratioI;
//					ray->tra = 0; }	} }
//		if ( flag[2] && ray->zp <= grid.zu[ray->kc - 1] ) {
//			ray->kc = ray->kc - 1;
//			if(ray->kc == 0) {
//				if(bdb==1)	{ray->kc = grid.km; ray->zp = Lz;}
//				else if(bdb==2) {
//					*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.bot - 1.0f )* ratioI;
//					ray->tra = 0;
//					break; } } }
//
//		// Do the same thing if it is happening in the positive direction
//		if ( !flag[0] && ray->xp >= grid.xu[ray->ic] ) {
//			ray->ic = ray->ic + 1;
//			if ( ray->ic == grid.im+1 )	{
//				if(bde==1)	{ray->ic = 1; ray->xp = 0;}
//				else if(bde==2) {
//#if adiabe == 1
//                                        t2         = (ray->jc-0.5f)*(jmax/grid.jm)+1.0f;
//                                        t3         = (ray->kc-0.5f)*(kmax/grid.km)+1.0f;
//					tf         = tex3D<myfloat>(tex_Tf,imax+1.0f,t2,t3);
//					pos        = (tf/Tnb[0] - Tnb[1]);
//					emiss.east = tex2D(tex_Ib,nb+0.5f ,pos+0.5f);
//#endif
//					*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.east - 1.0f )* ratioI;
//					ray->tra = 0;
//					break; } } }
//		if ( !flag[1] && ray->yp >= grid.yu[ray->jc] ) {
//			ray->jc = ray->jc + 1;
//			if ( ray->jc == grid.jm+1 )	{
//				if(bdn==1)	{ray->jc = 1; ray->yp = 0;}
//				else if(bdn==2) {
//					*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.north - 1.0f )* ratioI;
//					ray->tra = 0;
//					break; } } }
//		if ( !flag[2] && ray->zp >= grid.zu[ray->kc] ) {
//			ray->kc = ray->kc + 1;
//			if ( ray->kc == grid.km+1 )	{
//				if(bdt==1)	{ray->kc = 1; ray->zp = 0;}
//				else if(bdt==2) {
//					*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.top - 1.0f )* ratioI;
//					ray->tra = 0;
//					break; }  
//				else if(bdt==4) {
//                                        t1         = (ray->ic-0.5f)*(imax/grid.im)+1.0f;
//                                        t2         = (ray->jc-0.5f)*(jmax/grid.jm)+1.0f;
//					tf         = tex3D<myfloat>(tex_Tf,t1,t2,kmax+1.0f);
//					pos        = (tf/Tnb[0] - Tnb[1]);
//					emiss.top  = tex2D(tex_Ib, nb+0.5f ,pos+0.5f);
//					*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.west - 1.0f )* ratioI;
//					ray->tra = 0;
//					break; } } }
		// find the distance to cell boundary x,y,z -> the minimal will be the crossing
		myfloat dsx, dsy, dsz;
		dsx = ( grid.xu[ray->ic-flag[0]] - ray->xp ) / ray->sx;
		dsy = ( grid.yu[ray->jc-flag[1]] - ray->yp ) / ray->sy;
		dsz = ( grid.zu[ray->kc-flag[2]] - ray->zp ) / ray->sz;

		// interpolate to find the temperature of the cell (particle and fluid)
                t1 = (ray->ic-0.5f)*(imax/grid.im)+1.0f;
                t2 = (ray->jc-0.5f)*(jmax/grid.jm)+1.0f;
                t3 = (ray->kc-0.5f)*(kmax/grid.km)+1.0f;

		tf = tex3D<myfloat>(tex_Tf,t1,t2,t3);

		// interpolate to find the absorption of the cell (particle and fluid)
		pos = (tf/Tnb[0] - Tnb[1]);
		myfloat absco =  tex3D<myfloat>(*tex, nb+0.5f, pos+0.5f, g+0.5f);

		// black-body radiation of the cell (particle and fluid)
		myfloat blackpow = tex2D(tex_Ib,nb+0.5f ,pos+0.5f);
		myfloat ds = dsx;
		(void) ( (ds > dsy) && (ds = dsy) );
		(void) ( (ds > dsz) && (ds = dsz) );

		// update ray position and scattering length
		(void) ( (ds==dsx) && (ray->xp = grid.xu[ray->ic-flag[0]]) );
		(void) ( (ds!=dsx) && (ray->xp =   ray->xp + ds * ray->sx) );
		(void) ( (ds==dsy) && (ray->yp = grid.yu[ray->jc-flag[1]]) );
		(void) ( (ds!=dsy) && (ray->yp =   ray->yp + ds * ray->sy) );
		(void) ( (ds==dsz) && (ray->zp = grid.zu[ray->kc-flag[2]]) );
		(void) ( (ds!=dsz) && (ray->zp =   ray->zp + ds * ray->sz) );

		myfloat alpha  = 1.0f - __expf(-ds * (absco));

		*De_OERMc -= Ibmax * ray->tra * alpha * ratioI *
				( invTi4 * blackpow - 1.0f );

		// update transmissivity of the ray and total distance travelled
		ray->tra = ray->tra * (1-alpha);
		if ( ray->tra < toll)
		{
			*De_OERMc -= Ibmax * ray->tra * ratioI *
					( invTi4 * blackpow - 1.0f );
			ray->tra = 0;
		}

		// Updating cell indices and Boundary conditions if end is reached
		// efficient ray tracing method? Probably yes

		if ( dsx<dsy )
		{
			if ( dsx<dsz )
			{
				if ( flag[0] )
				{
					ray->ic = ray->ic-1;
					if ( ray->ic == 0 )
					{
						if(bdw==1)
						{
							ray->ic = grid.im;
							ray->xp = Lx;
						}
						else if(bdw==2)
						{       
#if adiabw == 1
                                                        t2         = (ray->jc-0.5f)*(jmax/grid.jm)+1.0f;
                                                        t3         = (ray->kc-0.5f)*(kmax/grid.km)+1.0f;
							tf         = tex3D<myfloat>(tex_Tf,1.0f,t2,t3);
							pos        = (tf/Tnb[0] - Tnb[1]);
							emiss.west = tex2D(tex_Ib, nb+0.5f ,pos+0.5f);
#endif
							*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.west - 1.0f )* ratioI;
							ray->tra = 0;
						}
						else if(bdw==3)
						{
							ray->xp = 0;
							ray->sx = -ray->sx;
							ray->ic = 1;
							flag[0] = (int)(ray->sx<0);
						}
					}
				}
				else
				{
					ray->ic = ray->ic+1;
					if ( ray->ic == grid.im+1 )
					{
						if(bde==1)
						{
							ray->ic = 1;
							ray->xp = 0;
						}
						else if(bde==2)
						{
#if adiabe == 1
                                                        t2         = (ray->jc-0.5f)*(jmax/grid.jm)+1.0f;
                                                        t3         = (ray->kc-0.5f)*(kmax/grid.km)+1.0f;
							tf         = tex3D<myfloat>(tex_Tf,imax+1.0f,t2,t3);
							pos        = (tf/Tnb[0] - Tnb[1]);
							emiss.east = tex2D(tex_Ib,nb+0.5f ,pos+0.5f);
#endif
							*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.east - 1.0f )* ratioI;
							ray->tra = 0;
						}
						else if(bde==3)
						{
							ray->xp = Lx;
							ray->sx = -ray->sx;
							ray->ic = grid.im;
							flag[0] = (int)(ray->sx<0);
						}
					}
				}
			}
			else
			{
				if ( flag[2] )
				{
					ray->kc = ray->kc-1;
					if ( ray->kc == 0 )
					{
						if(bdb==1)
						{
							ray->kc = grid.km;
							ray->zp = Lz;
						}
						else if(bdb==2)
						{
							*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.bot - 1.0f )* ratioI;
							ray->tra = 0;
						}
						else if(bdb==3)
						{
							ray->zp = 0;
							ray->sz = -ray->sz;
							ray->kc = 1;
							flag[3] = (int)(ray->sz<0);
						}
						else if(bdb==4)
						{       
                                                        t1         = (ray->ic-0.5f)*(imax/grid.im)+1.0f;
                                                        t2         = (ray->jc-0.5f)*(jmax/grid.jm)+1.0f;
							tf         = tex3D<myfloat>(tex_Tf,t1,t2,1.5f);
							pos        = (tf/Tnb[0] - Tnb[1]);
							emiss.bot  = tex2D(tex_Ib, nb+0.5f ,pos+0.5f);
							*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.bot - 1.0f )* ratioI;
							ray->tra = 0;
						}
						else if(bdb==5)
						{       
							ray->tra = 0;
             					}
					}
				}
				else
				{
					ray->kc = ray->kc+1;
					if ( ray->kc == grid.km+1 )
					{
						if(bdt==1)
						{
							ray->kc = 1;
							ray->zp = 0;
						}
						else if(bdt==2)
						{
							*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.top - 1.0f )* ratioI;
							ray->tra = 0;
						}
						else if(bdt==3)
						{
							ray->zp = Lz;
							ray->sz = -ray->sz;
							ray->kc = grid.km;
							flag[3] = (int)(ray->sz<0);
						}
						else if(bdt==4)
						{       
                                                        t1         = (ray->ic-0.5f)*(imax/grid.im)+1.0f;
                                                        t2         = (ray->jc-0.5f)*(jmax/grid.jm)+1.0f;
							tf         = tex3D<myfloat>(tex_Tf,t1,t2,kmax+0.5f);
							pos        = (tf/Tnb[0] - Tnb[1]);
							emiss.top  = tex2D(tex_Ib, nb+0.5f ,pos+0.5f);
							*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.top - 1.0f )* ratioI;
							ray->tra = 0;
						}
						else if(bdt==5)
						{       
							ray->tra = 0;
             					}
					}
				}
			}
		}
		else
		{
			if ( dsy<dsz )
			{
				if ( flag[1] )
				{
					ray->jc = ray->jc-1;
					if ( ray->jc == 0 )
					{
						if(bds==1)
						{
							ray->jc = grid.jm;
							ray->yp = Ly;
						}
						else if(bds==2)
						{
							*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.south - 1.0f )* ratioI;
							ray->tra = 0;
						}
						else if(bds==3)
						{
							ray->sy = 0;
							ray->sy = -ray->sy;
							ray->jc = 1;
							flag[2] = (int)(ray->sy<0);
						}
					}
				}
				else
				{
					ray->jc = ray->jc+1;
					if ( ray->jc == grid.jm+1 )
					{
						if(bdn==1)
						{
							ray->jc = 1;
							ray->yp = 0;
						}
						else if(bdn==2)
						{
							*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.north - 1.0f )* ratioI;
							ray->tra = 0;
						}
						else if(bdn==3)
						{
							ray->sy = Ly;
							ray->sy = -ray->sy;
							ray->jc = grid.jm;
							flag[2] = (int)(ray->sy<0);
						}
					}
				}
			}
			else
			{
				if ( flag[2] )
				{
					ray->kc = ray->kc-1;
					if ( ray->kc == 0 )
					{
						if(bdb==1)
						{
							ray->kc = grid.km;
							ray->zp = Lz;
						}
						else if(bdb==2)
						{
							*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.bot - 1.0f )* ratioI;
							ray->tra = 0;
						}
						else if(bdb==3)
						{
							ray->sz = 0;
							ray->sz = -ray->sz;
							ray->kc = 1;
							flag[3] = (int)(ray->sz<0);
						}
						else if(bdb==4)
						{       
                                                        t1         = (ray->ic-0.5f)*(imax/grid.im)+1.0f;
                                                        t2         = (ray->jc-0.5f)*(jmax/grid.jm)+1.0f;
							tf         = tex3D<myfloat>(tex_Tf,t1,t2,1.5f);
							pos        = (tf/Tnb[0] - Tnb[1]);
							emiss.bot  = tex2D(tex_Ib, nb+0.5f ,pos+0.5f);
							*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.bot - 1.0f )* ratioI;
							ray->tra = 0;
						}
						else if(bdb==5)
						{       
							ray->tra = 0;
             					}
					}
				}
				else
				{
					ray->kc = ray->kc+1;
					if ( ray->kc == grid.km+1 )
					{
						if(bdt==1)
						{
							ray->kc = 1;
							ray->zp = 0;
						}
						else if(bdt==2)
						{
							*De_OERMc -= Ibmax * ray->tra *( invTi4 * emiss.top - 1.0f )* ratioI;
							ray->tra = 0;
						}
						else if(bdt==3)
						{
							ray->sy = Lz;
							ray->sz = -ray->sz;
							ray->kc = grid.km;
							flag[3] = (int)(ray->sz<0);
						}
						else if(bdt==4)
						{       
                                                        t1         = (ray->ic-0.5f)*(imax/grid.im)+1.0f;
                                                        t2         = (ray->jc-0.5f)*(jmax/grid.jm)+1.0f;
							tf         = tex3D<myfloat>(tex_Tf,t1,t2,kmax+0.5f);
							pos        = (tf/Tnb[0] - Tnb[1]);
							emiss.top  = tex2D(tex_Ib, nb+0.5f ,pos+0.5f);
							*De_OERMc -= Ibmax * ray->tra*( invTi4 * emiss.top - 1.0f )* ratioI;
							ray->tra = 0;
						}
						else if(bdt==5)
						{       
							ray->tra = 0;
             					}
					}
				}
			}
		}
		counter+=1;

	}  // closing the while statement
}

extern "C" void mc_gpu_(myfloatF *Tfort, int *zstart, int *rank)
{

	myfloat kappamax,Tmax;

	/**************************************************************/
	/********** CREATING GRID AND TEMPERATURE FIELD ***************/
	/**************************************************************/

	Var_CPU varCPU[grid_num];
	Gridn gridCPU[grid_num];
	for (int grd = 0; grd < grid_num; grd++)
	{
		varCPU[grd].mk_grid(maxi[grd],maxj[grd],maxk[grd],maxi[0]);
		gridCPU[grd].mk_grid(maxi[grd],maxj[grd],maxk[grd],maxi[0],maxs[grd]);
	}
	for (int k = 0; k < (kmax+2); k++)
	{
		for (int j = 0; j < (jmax+2); j++)
		{
			for (int i = 0; i < (imax+2); i++)
			{
				varCPU[0].T[idx_T(i,j,k,imax,jmax)] = (myfloat) Tfort[idx_T(i,j,k,imax,jmax)];
                                //if(varCPU[0].T[idx_T(i,j,k,imax,jmax)]<490 || varCPU[0].T[idx_T(i,j,k,imax,jmax)]>1000)
 				// if(i<=imax && i>=1)
 				// if(j<=jmax && j>=1)
 				// if(k<=kmax && k>=1)
 				//  printf("WTF here %d %d %d %f\n",i,j,k,Tfort[idx_T(i,j,k,imax,jmax)]);
			}
		}
	}

				
	// interpolating temperature and finding new concentration on coarser grid
	// interp3D(varCPU);
 
	Tmax = 0;
	for (int k=1; k<kmax+1; k++)
	{
		for (int j=1; j<jmax+1; j++)
		{
			for (int i=1; i<imax+1; i++)
			{
				Tmax = MAX(Tmax,varCPU[0].T[idx_T(i,j,k,maxi[0],maxj[0])]);
			}
	                Tmax = MAX(Tmax,
                                   (varCPU[0].T[idx_T(0,j,k,maxi[0],maxj[0])]+varCPU[0].T[idx_T(1,j,k,maxi[0],maxj[0])])*0.5);
	                Tmax = MAX(Tmax,
                                   (varCPU[0].T[idx_T(imax,j,k,maxi[0],maxj[0])]+varCPU[0].T[idx_T(imax+1,j,k,maxi[0],maxj[0])])*0.5);
		}
	}

	/**************************************************************/
	/***********  READ THE TABLES *********************************/
	/**************************************************************/

	NarrowBand *narrBand;
	myfloat *Tnb;
	myfloat *prob_h, *probg_h, *prob_h2;

	prob_h   = (myfloat*)malloc(nB*nT   *sizeof(myfloat));
	probg_h  = (myfloat*)malloc(nB*nT*nQ*sizeof(myfloat));
	prob_h2  = (myfloat*)malloc(nB*2    *sizeof(myfloat));

	narrBand = (NarrowBand*)malloc(nB *  sizeof(NarrowBand));
	Tnb = (myfloat*)malloc(nT*sizeof(myfloat));

	readT(narrBand, Tnb, kP, prob_h, probg_h, Tmax, &kappamax);

#if ((adiabw ==1) ||(adiabe == 1)) && (calcwalls == 1)
	myfloat *prob_hs;
        prob_hs = (myfloat*)malloc(nB*2*sizeof(myfloat)); 
	readS(narrBand, Tnb, Tmax, prob_hs); 
#endif

	/**************************************************************/
	/***********  FINISHED READING ********************************/
	/**************************************************************/

	//Sorting NarrowBands based on kavg of the band
#if srt == 1
	sort_idx(narrBand);
#endif

	/**************************************************************/
	/************* MEMORY COPY TO THE GPU's ***********************/
	/**************************************************************/


	for(int gpu = 0; gpu < num_gpu; gpu++) {
		hipSetDevice(gpu);
//                printf("we are in rank %d setting device number %d\n",*rank,gpu);
//		cudaCheckErrors2("set device",*rank);
	        hipStream_t streams[num_streams];
		cudaCheckErrors("Before streams");
		for (int i=0; i<num_streams; i++)
		{
			hipStreamCreate(&streams[i]);

		}

		// grid copy to GPU
//		cudaCheckErrors("Before malloc");
		hipMalloc((void**)&gridGPU[gpu], grid_num * sizeof(Gridn));
//		cudaCheckErrors("Before grid copy");
		grid_copy(gridCPU, gridGPU[gpu]);
		// Wall emission copy to GPU
		hipMalloc((void**)&Ibw_d[gpu], nB * sizeof(EmissSpec));
		black_copy(Ibw_d[gpu],narrBand);

		myfloat Ib[nT][nB];

		for(int t = 0; t<nT; t++ )
			for(int nb = 0; nb<nB; nb++ )
			{
#if srt ==1
				Ib[t][nb] = I_blackC(Tnb[t],narrBand[narrBand[nb].idx].wvc);
#else
				Ib[t][nb] = I_blackC(Tnb[t],narrBand[nb].wvc);
#endif
			}

		// Copy surface probability and blackbody emission
		// Create explicit channel description (could use an implicit as well)
		hipChannelFormatDesc DescIb = hipCreateChannelDesc<myfloat>();
		hipMallocArray(&cuArrayI[gpu], &DescIb, nB, nT);
		hipMemcpyToArray(cuArrayI[gpu], 0, 0, Ib, nB*nT*sizeof(myfloat), hipMemcpyHostToDevice);
		tex_Ib.addressMode[0] = hipAddressModeClamp;
		tex_Ib.addressMode[1] = hipAddressModeClamp;
		tex_Ib.filterMode = hipFilterModeLinear;
		tex_Ib.normalized = false;
		hipBindTextureToArray(tex_Ib, cuArrayI[gpu], DescIb);

		//textured memory copy of interpolated temperature
		hipMalloc((void**)&tex_tempf_d[gpu], sizeof(hipTextureObject_t) );
		temp_fluid_copy(tex_tempf_d[gpu], varCPU);

		//memory copy of -> sorted index		    idx_d
		//				 -> central wavenumber 		wvc_d
		//				 -> discrete temperature	Tnb_d
		//				 -> textured asb coeff		tex_d
		//				 -> textured emiss prob		tex_prob_d
		//				 -> textured surfa prob		tex_prob_s
		hipMalloc((void**)&idx_d[gpu],nB*sizeof(int));
		hipMalloc((void**)&wvc_d[gpu],nB*sizeof(myfloat));
		hipMalloc((void**)&Tnb_d[gpu],nT*sizeof(myfloat));
		hipMalloc((void**)&tex_d[gpu],   sizeof(hipTextureObject_t) );
		hipMalloc((void**)&tex_prob_d[gpu], sizeof(hipTextureObject_t) );
		hipMalloc((void**)&tex_prob_s[gpu], sizeof(hipTextureObject_t) );
		narrowband_copy(narrBand, wvc_d[gpu], idx_d[gpu], tex_d[gpu], tex_prob_d[gpu], Tnb_d[gpu], Tnb);



	        hiprandDirectionVectors32_t *d_rngVectors;
	        hipMalloc((void **)&d_rngVectors, 2 * sizeof(hiprandDirectionVectors32_t));
	        hiprandDirectionVectors32_t *rngDirections;
	        hiprandGetDirectionVectors32(&rngDirections, HIPRAND_DIRECTION_VECTORS_32_JOEKUO6);
	        hipMemcpy(d_rngVectors, rngDirections, 2*sizeof(hiprandDirectionVectors32_t), hipMemcpyHostToDevice);

		// CUDA memory allocation
		int tm = (int) ((Tmax - Tnb[0])/(Tnb[1]-Tnb[0]));
		for(int j = 0; j < 2; j++)
			for(int i = 0; i < nB; i++ )
			{
				prob_h2[idx_p(j,i)] = prob[tm+j][i];
			}
		hipMemcpyToSymbol(HIP_SYMBOL(probd), prob_h2 , nB*2*sizeof(myfloat) ,0,hipMemcpyHostToDevice);
#if ((adiabw==1) || (adiabe==1)) && (calcwalls == 1)
		hipMemcpyToSymbol(HIP_SYMBOL(probs), prob_hs , nB*2*sizeof(myfloat) ,0,hipMemcpyHostToDevice);
#endif
		cudaCheckErrors("Malloc fail");

		/**************************************************************/
		/***********  STARTING CUDA ROUTINES **************************/
		/**************************************************************/

		/**************************************************************/
		/***************** FLUID MONTE CARLO **************************/
		/**************************************************************/

		int block_num, block_numS, thread, threadS;
                block_num  = MIN(N /blockdim,nblocks);
                block_numS = MIN(NS/blockdim,nblocks);
                if(block_num<1)  {block_num =1; thread = N; }
                else              thread = blockdim;
                if(block_numS<1) {block_numS=1; threadS= NS; }
                else              threadS = blockdim;

		for (int i=0; i<num_streams; i++)
		{
			kernel_fluid<<<block_num   , thread , 0, streams[i]>>>(d_rngVectors, gridGPU[gpu], wvc_d[gpu], Tnb_d[gpu], N, num_streams, i, kappamax, Tmax, idx_d[gpu],
					tex_tempf_d[gpu], tex_d[gpu], tex_prob_d[gpu], gpu, *zstart, Ibw_d[gpu]);
                }
#if ((adiabw ==1) ||(adiabe == 1)) && (calcwalls == 1)
		for (int i=0; i<num_streams; i++)
		{
			kernel_surface<<<block_numS, threadS, 0, streams[i]>>>(gridGPU[gpu], wvc_d[gpu], Tnb_d[gpu], NS, num_streams, i, Tmax, idx_d[gpu],
					tex_tempf_d[gpu], tex_d[gpu], tex_prob_d[gpu], gpu, *zstart);
		}
#endif 
		cudaCheckErrors("Failed kernel execution");
	}

	//freeing all the CPU used variables (GPU is freed automatically by hipDeviceReset(); )
	for(int grd=0; grd<grid_num; grd++)
	{
		varCPU[grd].destroyVar();
		gridCPU[grd].destroyVar();
	}
	free(prob_h);
	free(probg_h);
	free(prob_h2);
	free(narrBand);
	free(Tnb);
}

extern "C" void get_results_(myfloatF resfort[(imax+2)*(jmax+2)*(kmax/p_col+2)], myfloatF varfort[(imax+2)*(jmax+2)*(kmax/p_col+2)], int *rank)
{

        time_t timer;
        char buffer[26];
        struct tm* tm_info;

	myfloat *host[num_streams][num_gpu];
	myfloat *varh[num_streams][num_gpu];
	myfloat *hosS[num_streams][num_gpu];
	myfloat *hvrS[num_streams][num_gpu];
	for (int i=0; i<num_streams; i++) {
		for(int gpu = 0; gpu < num_gpu; gpu++) {
			host[i][gpu] = (myfloat*)malloc(N   *sizeof(myfloat));
			varh[i][gpu] = (myfloat*)malloc(N   *sizeof(myfloat));
			hosS[i][gpu] = (myfloat*)malloc(2*NS*sizeof(myfloat));
	                hvrS[i][gpu] = (myfloat*)malloc(2*NS*sizeof(myfloat));
		}
	}
	myfloat *device[num_streams][num_gpu];
	myfloat *vard[num_streams][num_gpu];
	myfloat *devS[num_streams][num_gpu];
	myfloat *varS[num_streams][num_gpu];

	for(int gpu = 0; gpu < num_gpu; gpu++) {
		hipSetDevice(gpu);
	        hipStream_t streams[num_streams];
		hipDeviceSynchronize();
		cudaCheckErrors("Malloc fail synchronize");
		for (int i=0; i<num_streams; i++)
		{
			hipMalloc((void**)&device[i][gpu],N * sizeof(myfloat));
			hipMalloc((void**)&vard[i][gpu]  ,N * sizeof(myfloat));
			hipMalloc((void**)&varS[i][gpu]  ,2*NS * sizeof(myfloat));
			hipMalloc((void**)&devS[i][gpu]  ,2*NS * sizeof(myfloat));
			cudaCheckErrors("Malloc fail device");
			hipStreamCreate(&streams[i]);
		}


		for (int i=0; i<num_streams; i++)
		{
			kernel_results<<<nblocks, blockdim, 0, streams[i]>>>(vard[i][gpu],device[i][gpu],varS[i][gpu],devS[i][gpu],N,NS,i);
		}
		cudaCheckErrors("Failed kernel execution");
	}
	for(int gpu = 0; gpu < num_gpu; gpu++) {
		hipSetDevice(gpu);
	        hipStream_t streams[num_streams];
		for (int i = 0; i < num_streams; i++)
		{
			hipStreamCreate(&streams[i]);
			hipMemcpyAsync(host[i][gpu],device[i][gpu],N *sizeof(myfloat),hipMemcpyDeviceToHost,streams[i]);
			hipMemcpyAsync(varh[i][gpu],vard[i][gpu]  ,N *sizeof(myfloat),hipMemcpyDeviceToHost,streams[i]);
#if ((adiabw == 1) || (adiabe == 1)) && (calcwalls == 1)
			hipMemcpyAsync(hosS[i][gpu],devS[i][gpu],2*NS*sizeof(myfloat),hipMemcpyDeviceToHost,streams[i]);
			hipMemcpyAsync(hvrS[i][gpu],varS[i][gpu],2*NS*sizeof(myfloat),hipMemcpyDeviceToHost,streams[i]);
#endif
			cudaCheckErrors("Cuda memory copy asynchronous, device to host");
		}
		cudaCheckErrors("Copying to host fail");
	}

	cudaCheckErrors("unbind and/or free fail");

	/**************************************************************/
	/***************** RESETTING DEVICE MEMORY ********************/
	/**************************************************************/


	for(int gpu = 0; gpu < num_gpu; gpu++)
	{
		hipSetDevice(gpu);

		hipFree(Ibw_d[gpu]);
		hipFree(gridGPU[gpu]);
		hipFree(tex_tempf_d[gpu]);
		hipFree(idx_d[gpu]);
		hipFree(wvc_d[gpu]);
		hipFree(Tnb_d[gpu]);
		hipFree(tex_d[gpu]);
		hipFree(tex_prob_d[gpu]);
		hipFreeArray(cuArrayI[gpu]);
		hipFreeArray(cuArrayP[gpu]);
		cudaCheckErrors("unbind and/or free fail");

		for(int i=0; i<num_streams; i++) {
			hipFree(device[i][gpu]);
			hipFree(vard[i][gpu]);
			hipFree(devS[i][gpu]);
			hipFree(varS[i][gpu]);
		}
		hipDeviceReset();
	}

	/**************************************************************/
	/************ RETURNING RESULTS IN A 3D FASHION ***************/
	/**************************************************************/

	int i, j, k;
	for(int stream = 0; stream < num_streams; stream++) {
		for(int gpu = 0; gpu < num_gpu; gpu++) {
			for(int n = 0; n < N; n++) {
				i = (n / (kmax*jmax/num_gpu/p_col) + 1) + (stream)*imax/num_streams;
				j = (n / (kmax/p_col) + 1 - (i-1-(stream)*imax/num_streams)*jmax/num_gpu);
				k = (n - (kmax/p_col) * (j-1 + (i-1-(stream)*imax/num_streams)*jmax/num_gpu) + 1);
                                j+= gpu*jmax/num_gpu;
				resfort[idx_F(i,j,k)] = (myfloatF) host[stream][gpu][n];
				varfort[idx_F(i,j,k)] = (myfloatF) powf(varh[stream][gpu][n],0.5);
			}
#if (adiabw==1) || (adiabe==1)
			for(int n = 0; n < NS; n++) {
				j = n / (kmax/p_col) + 1;
				k = n - (kmax/p_col) * (j - 1) + 1;
				j+= (stream+num_streams*gpu)*(jmax/(num_gpu*num_streams)); 
#if calcwalls==1
				resfort[idx_F(0,j,k)] = (myfloatF) hosS[stream][gpu][n];
				varfort[idx_F(0,j,k)] = (myfloatF) powf(hvrS[stream][gpu][n],0.5);
				resfort[idx_F(imax+1,j,k)] = (myfloatF) hosS[stream][gpu][NS+n];
				varfort[idx_F(imax+1,j,k)] = (myfloatF) powf(hvrS[stream][gpu][NS+n],0.5);
#else
				resfort[idx_F(0,j,k)] = (myfloatF) 0.0f; 
				varfort[idx_F(0,j,k)] = (myfloatF) 0.0f;
				resfort[idx_F(imax+1,j,k)] = (myfloatF) 0.0f;
				varfort[idx_F(imax+1,j,k)] = (myfloatF) 0.0f;
#endif
			}
#endif
		}
	}

	for (int i=0; i<num_streams; i++) {
		for(int gpu = 0; gpu < num_gpu; gpu++) {
			free(host[i][gpu]);
			free(varh[i][gpu]);
		}
	}
        time(&timer);
        tm_info = localtime(&timer);
    
//	  printf("Finished gpu routine in rank %d  \t",*rank);
//        strftime(buffer, 26, "%Y-%m-%d %H:%M:%S", tm_info);
//        puts(buffer);


}

__device__ __forceinline__ void kernel_find(Count *count, myfloat *Tnb, myfloat Tmax, hiprandState_t *state, hipTextureObject_t *tex_prob)
{
	int tm = (int) ((Tmax - Tnb[0])/(Tnb[1]-Tnb[0]));

	int nb = 0;
	int g = 0;
	for (int h = 0; h<nB; h++)
	{
		count->nb_cnt[h] = 0;
		for (int f = 0; f<nQ; f++)
		{
			count->g_cnt[f][h] = 0;
		}
	}
	/***********  ENTERING THE PHOTON LOOP ****************************/
	for (int h=0; h<photmax; h++)
	{
		// now we have to define the absorption narrow band wavenumber and the
		// quadrature point from prob and probg
		find_band (Tnb,Tmax,&g,&nb,state,tm,tex_prob);
		count->nb_cnt[nb] += 1;
		count->g_cnt[g][nb] +=1;

		/***********  OUT OF THE VARIANCE LOOP ****************************/
	}

}
__device__ __forceinline__ void kernel_findS(Count *count, myfloat *Tnb, myfloat Tmax, hiprandState_t *state)
{
	int tm = (int) ((Tmax - Tnb[0])/(Tnb[1]-Tnb[0]));

	int nb = 0;
	int g = 0;
	for (int h = 0; h<nB; h++)
	{
		count->nb_cnt[h] = 0;
		for (int f = 0; f<nQ; f++)
		{
			count->g_cnt[f][h] = 0;
		}
	}
	/***********  ENTERING THE PHOTON LOOP ****************************/
	for (int h=0; h<photmax; h++)
	{
		// now we have to define the absorption narrow band wavenumber and the
		// quadrature point from prob and probg
		find_bandS(Tnb,Tmax,&g,&nb,state,tm);
		count->nb_cnt[nb] += 1;
		count->g_cnt[g][nb] +=1;

		/***********  OUT OF THE VARIANCE LOOP ****************************/
	}

}
__device__ __forceinline__ void emiss_ang(Beam *ray, hiprandState_t *state)
{
	myfloat phi   = hiprand_uniform(state)*2*pi;
	myfloat theta = acosf( 1 - 2*hiprand_uniform(state) );
	ray->sx = __cosf(theta);
	ray->sy = __sinf(theta)*__cosf(phi);
	ray->sz = __sinf(theta)*__sinf(phi);
}
__device__ __forceinline__ void emiss_angSobol(Beam *ray, hiprandStateSobol32_t *state1, hiprandStateSobol32_t *state2)                                                                                     
{
          myfloat phi   = hiprand_uniform(state1)*2*pi;
          myfloat theta = acosf( 1 - 2*hiprand_uniform(state2) );
          ray->sx = __cosf(theta);
          ray->sy = __sinf(theta)*__cosf(phi);
          ray->sz = __sinf(theta)*__sinf(phi);
}
__device__ __forceinline__ void emiss_angS(Beam *ray, hiprandState_t *state, int bound)
{
	myfloat phi   = hiprand_uniform(state)*2*pi;
	myfloat theta;
        theta = acosf( 1.0f - 2*hiprand_uniform(state) )/2;
        if(bound==1) theta = theta+pi/2.0f;
	ray->sx = __cosf(theta);
	ray->sy = __sinf(theta)*__cosf(phi);
	ray->sz = __sinf(theta)*__sinf(phi);
}
__device__ __forceinline__ void find_band(myfloat *Tnb, myfloat Tmax, int *g, int *nb, hiprandState_t *state, int tm, hipTextureObject_t *tex_prob)
{

	myfloat Rwave = hiprand_uniform(state);
	// find index of temperature
	//temperature index is t and t+1, now search for R on t and t+1
	int t;
	if( (Tmax - Tnb[tm]) < (Tnb[tm+1] - Tmax) )
	{
		t = tm;
		int nb1 = 0;
		int w = nB;
		while (w > nb1+1)
		{
			int d = (nb1+w)/2;
			(void)((Rwave <  probd[t-tm][d] ) && (w=d));
			(void)((Rwave >= probd[t-tm][d] ) && (nb1=d));
		};
		nb1 = nb1+1;
		if( Rwave <  probd[t-tm][0] )
			nb1 = 0;
		if(Rwave == 1 )
			nb1 = nB -1;

		*nb = nb1;
	}
	else
	{
		t = tm+1;
		int nb2 = 0;
		int w = nB;
		while (w > nb2+1)
		{
			int d = (nb2+w)/2;
			(void)((Rwave <  probd[t-tm][d] ) && (w=d));
			(void)((Rwave >= probd[t-tm][d] ) && (nb2=d));
		};
		nb2 = nb2+1;
		if(Rwave <  probd[t-tm][0] )
			nb2 = 0;
		if(Rwave == 1 )
			nb2 = nB -1;
		*nb = nb2;
	}

#if grey == 1
	*g=0;
#else
	myfloat Rwave2 = hiprand_uniform(state);
	myfloat prob;

	*g =0;
	int w = nQ;
	while (w > *g+1)
	{
		int d = (*g+w)/2;
		prob = tex3D<myfloat>(*tex_prob, d+0.5f, *nb+0.5f, t+0.5f);
		(void)((Rwave2 < prob ) && ( w=d));
		(void)((Rwave2 >= prob ) && (*g=d));
	};
	*g = *g+1;
	prob = tex3D<myfloat>(*tex_prob, 0.5f, *nb+0.5f, t+0.5f);
	(void)((Rwave2 < prob ) && (*g = 0));
	(void)((Rwave2 == 1)    && (*g = nQ-1));
#endif

}
__device__ __forceinline__ void find_bandS(myfloat *Tnb, myfloat Tmax, int *g, int *nb, hiprandState_t *state, int tm)
{

	myfloat Rwave = hiprand_uniform(state);
	// find index of temperature
	//temperature index is t and t+1, now search for R on t and t+1
        int t;
        if( (Tmax - Tnb[tm]) < (Tnb[tm+1] - Tmax) )
        {
                t = tm; 
                int nb1 = 0;
                int w = nB;
                while (w > nb1+1)
                {
                        int d = (nb1+w)/2;
                        (void)((Rwave <  probs[t-tm][d] ) && (w=d));
                        (void)((Rwave >= probs[t-tm][d] ) && (nb1=d));
                };
                nb1 = nb1+1; 
                if( Rwave <  probd[t-tm][0] )
                        nb1 = 0;
                if(Rwave == 1 )
                        nb1 = nB -1;

                *nb = nb1;
        }
        else
        {
                t = tm+1; 
                int nb2 = 0;
                int w = nB;
                while (w > nb2+1)
                {
                        int d = (nb2+w)/2;
                        (void)((Rwave <  probs[t-tm][d] ) && (w=d));
                        (void)((Rwave >= probs[t-tm][d] ) && (nb2=d));
                };
                nb2 = nb2+1;
                if(Rwave <  probd[t-tm][0] )
                        nb2 = 0;
                if(Rwave == 1 )
                        nb2 = nB -1;
                *nb = nb2;
        }
#if grey == 1
	*g = 0;
#else
	myfloat Rwave2 = hiprand_uniform(state);

	*g =0;
	int w = nQ;
	while (w > *g+1)
	{
		int d = (*g+w)/2;
		(void)((Rwave2 <  probq[d] ) && ( w=d));
		(void)((Rwave2 >= probq[d] ) && (*g=d));
	};
	*g = *g+1;
	(void)((Rwave2 < probq[0] ) && (*g = 0));
	(void)((Rwave2 == 1)        && (*g = nQ-1));

#endif

}
__device__ __forceinline__ void wave_find(int *nb, int *g, int *countnb, int *countg, int *idx_nb, Count *cnt)
{
	int tmp=1;
	while(tmp)
	{
		if(*countnb < cnt->nb_cnt[idx_nb[*nb]])
		{
			*countnb+=1;
			tmp=0;
		}
		else
		{
			*nb +=1;
			*countnb=0;
			*countg=0;
			*g=0;
			tmp=1;
		}
	};
	while(!tmp)
	{
		if(*countg < cnt->g_cnt[*g][idx_nb[*nb]])
		{
			*countg+=1;
			tmp=1;
		}
		else
		{
			*g +=1;
			*countg=0;
			tmp=0;
		}
	};
}


struct PAIR
{
	int pos;
	myfloat val;
};

bool compare(PAIR p1, PAIR p2) {return p1.val < p2.val;}

void sort_idx(NarrowBand *narrBand)
{
	vector<PAIR> p(nB);

	for (int nb = 0; nb<nB; nb++)
	{
		p[nb].pos = nb;
		p[nb].val = narrBand[nb].kavg;
	}

	sort( p.begin(), p.end(), compare );

	for (int nb = 0; nb<nB; nb++)
	{
		narrBand[nb].idx = p[nb].pos;
	}
}

__global__ void kernel_results(myfloat *var, myfloat *device, myfloat *varS, myfloat *devS, int n, int ns, int stream)
{

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	volatile int tx = threadIdx.x;

	for (int idx = tid; idx < n ; idx += blockDim.x * gridDim.x)
	{
		device[idx] = solution[stream][idx];
		var[idx]    = variance[stream][idx];
	}
	for (int idx = tid; idx < ns; idx += blockDim.x * gridDim.x)
	{
		devS[idx]     = s_surf[stream][idx][0];
		varS[idx]     = v_surf[stream][idx][0];
		devS[ns+idx]  = s_surf[stream][idx][1];
		varS[ns+idx]  = v_surf[stream][idx][1];
	}
}

__global__ void kernel_surface(Gridn *my_grid, myfloat *wvc_d, myfloat *Tnb_t, int n, int ns, int stream, myfloat Tmax, int *idx_nb,
		hipTextureObject_t *tex_Tf, hipTextureObject_t *tex, hipTextureObject_t *tex_prob,
		int gpu, int zstart)
{
	Count cnt;

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	volatile int tx = threadIdx.x;

	// for the moment ray stays in the register memory, if it spills,
	// we have to move it to the shared memory (it should fit in register memory)
	Beam ray;
	// we start defining the ray-Ibmax that is equal for every position

	Emission emiss; //It is not going to be used since this is done only for adiabatic walls

	__const__ myfloat Tnb[] = {Tnb_t[1] - Tnb_t[0], (Tnb_t[0])/(Tnb_t[1] - Tnb_t[0])};
	myfloat invTi4, Ibmax, ratioI;
	myfloat pos;

	//define a narrow band
	myfloat wvc;
	__shared__ hiprandState_t state[blockdim];
	// this is basically the loop over the indices in the CPU.
	// the indices are found in relation with the stream-block-Thread design.
	// this is a grid-stride loop, hence every thread will do multiple cells

	for (int bound = boundini; bound < boundend; bound++) {
		/* bound = 0 is the emission from the west surface (i=0)
		 * bound = 1 is the emission from the east surface (i=imax)
		 */
		hiprand_init(1234, (bound+1)*(tid+1)*(stream+1)*(1+gpu)*zstart, 0, &state[tx]);  // 	Initialize CURAND
#if srt == 1
		kernel_findS(&cnt, Tnb_t, Tmax, &state[tx]);
#endif
                for (int idx  = tid; idx < n; idx += blockDim.x * gridDim.x)
		{
			s_surf[stream][idx][bound] = 0;
			v_surf[stream][idx][bound] = 0;

			ray.j = idx / (kmax/p_col) + 1;
			ray.k = idx - (kmax/p_col) * (ray.j - 1) + zstart;
                        ray.j+= (stream+ns*gpu)*(jmax/(num_gpu*ns));
			if(bound==0) {
				ray.i = 1;
				ray.Ti = tex3D<myfloat>(*tex_Tf, 1.0f, ray.j+0.5f, ray.k+0.5f);
				Ibmax  = epsw * stefan * pow4(Tmax)/photmax;}
			else		 {
				ray.i = imax;
				ray.Ti = tex3D<myfloat>(*tex_Tf, ray.i+1.0f, ray.j+0.5f, ray.k+0.5f);
				Ibmax  = epse * stefan * pow4(Tmax)/photmax;}
			myfloat De_var[nVar];

			for (int v = 0; v<nVar; v++)
			{
				De_var[v] = 0;
				int g=0;
				int nb=0;

				int countnb=0;
				int countg=0;

				myfloat De_OERMc=0;
				/***********  ENTERING THE PHOTON LOOP, DEVIDED INTO SUB-BUNDLES *****************/

				for (int h=0; h<photmax; h++)
				{
					if(bound==0) 	 ray.xp = 0;
					else		 ray.xp = Lx;
					ray.yp = my_grid[0].y[ray.j];
					ray.zp = my_grid[0].z[ray.k];
					ray.ic = ray.i;
					ray.jc = ray.j;
					ray.kc = ray.k;
					int flag[3];
					// Now we have to deal with random numbers, selecting the angles
					// define the scattering function
					emiss_angS(&ray, &state[tx], bound);

					if(fabsf(ray.sx) <= 1e-10)
					        ray.sx = (1-2*bound)*1e-10;
					if(fabsf(ray.sy) <= 1e-10)
					        ray.sy = ((ray.sy > 0) - (ray.sy <= 0))*1e-10;
					if(fabsf(ray.sz) <= 1e-10)
						ray.sz = ((ray.sz > 0) - (ray.sz <= 0))*1e-10;
					flag[0] = (int)(ray.sx<0);
					flag[1] = (int)(ray.sy<0);
					flag[2] = (int)(ray.sz<0);

					//Find the narrowband index and the g index, based on random or based on the previous count
#if srt == 1
					wave_find(&nb,&g,&countnb,&countg,idx_nb,&cnt);
#else
					// non sorted monte carlo
					int tm =  (int) ((Tmax - Tnb_t[0])/(Tnb_t[1]-Tnb_t[0]));
					find_bandS(Tnb_t,Tmax,&g,&nb,&state[tx],tm);
#endif
					// the ray emanating from cell i,j,k is completely defined now
					// now we have to march it
					// initializing transmissivity

					ray.tra = 1.0f;
					wvc  	= wvc_d[nb];

	                                // Calculate parameters for calculation of De_OERMc in march_ray
	                                pos = ((ray.Ti - Tnb_t[0])/(Tnb_t[1]-Tnb_t[0]));
	                                invTi4 = 1.0f / tex2D(tex_Ib,nb+0.5f ,pos+0.5f);
	                                pos = ((Tmax - Tnb_t[0])/(Tnb_t[1]-Tnb_t[0]));
	                                ratioI = (1.0f / invTi4) / tex2D(tex_Ib,nb+0.5f ,pos+0.5f);

					// Adjust the cells according to the scattering of the ray
					// if the ray originates from the negative cell boundary and it
					// is going in the negative direction, shift cells
					if ( flag[0] && ray.xp == my_grid[0].xu[ray.ic - 1] )
						ray.ic = ray.ic - 1;
					if ( flag[1] && ray.yp == my_grid[0].yu[ray.jc - 1] )
						ray.jc = ray.jc - 1;
					if ( flag[2] && ray.zp == my_grid[0].zu[ray.kc - 1] )
						ray.kc = ray.kc - 1;
					// Do the same thing if it is happening in the positive direction
					if ( !flag[0] && ray.xp == my_grid[0].xu[ray.ic] )
						ray.ic = ray.ic + 1;
					if ( !flag[1] && ray.yp == my_grid[0].yu[ray.jc] )
						ray.jc = ray.jc + 1;
					if ( !flag[2] && ray.zp == my_grid[0].zu[ray.kc] )
						ray.kc = ray.kc + 1;

					/**************************************************************************************/
					/************************** ENTERING THE MARCHING LOOP ********************************/
					/**************************************************************************************/

					//loop on all grids with maximum counter my_grid[grd].sm
					for(int grd = 0; grd < grid_num; grd++)
					{
						march_ray(&ray, nb, g, flag, &De_OERMc, Ibmax, my_grid[grd], ratioI, invTi4,
								*tex_Tf    , emiss, wvc, Tnb, tex, tollS);
						if(grd<grid_num-1) {
							//adapting grid
							ray.ic += my_grid[grd].im/my_grid[grd+1].im-1;
							ray.jc += my_grid[grd].jm/my_grid[grd+1].jm-1;
							ray.kc += my_grid[grd].km/my_grid[grd+1].km-1;
							ray.ic /= my_grid[grd].im/my_grid[grd+1].im;
							ray.jc /= my_grid[grd].jm/my_grid[grd+1].jm;
							ray.kc /= my_grid[grd].km/my_grid[grd+1].km;
						}
					};

//			 if((bound==1)&&(ray.j==45)&&(ray.k==861)) 
//                          printf("Finished ray, photon: %d direc: %f %f %f E: %f, finished in %d %d %d\n",h,ray.sx,ray.sy,ray.sz,De_OERMc,ray.ic,ray.jc,ray.kc);
				}

				/***********  OUT OF THE PHOTON LOOP ****************************/
				s_surf[stream][idx][bound] += De_OERMc;
				De_var[v] = De_OERMc;
			}
			/***********  OUT OF THE VARIANCE LOOP ****************************/

			s_surf[stream][idx][bound] /= nVar;
			v_surf[stream][idx][bound] = 0;
			for (int v = 0; v<nVar; v++)
			{
				v_surf[stream][idx][bound] += 1./(nVar-0.5) * 1./(nVar-1) * (De_var[v] - s_surf[stream][idx][bound]) * (De_var[v] - s_surf[stream][idx][bound]);
			}
		}
		/***********  OUT OF THE GRID-STRIDE LOOP ****************************/
	} //out of the boundary loop
}
