#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "../param/param.h"
#include "definitions.h"

#include <iostream>
#include <vector>
#include <algorithm>
#include "memory.h"

extern myfloat probg[nT][nB][nQ];

void black_copy(EmissSpec *Ibw_d, NarrowBand *narrBand)
{

	EmissSpec *Ibw_h;
	Ibw_h = (EmissSpec*)malloc(sizeof(EmissSpec));
	Ibw_h->allocVar(nB);

    for(int nb = 0; nb<nB; nb++ )
    {
#if srt ==1
    	Ibw_h->west[nb]  = I_blackC(Tww,narrBand[narrBand[nb].idx].wvc);
    	Ibw_h->east[nb]  = I_blackC(Twe,narrBand[narrBand[nb].idx].wvc);
    	Ibw_h->north[nb] = I_blackC(Twn,narrBand[narrBand[nb].idx].wvc);
    	Ibw_h->south[nb] = I_blackC(Tws,narrBand[narrBand[nb].idx].wvc);
    	Ibw_h->top[nb]   = I_blackC(Twt,narrBand[narrBand[nb].idx].wvc);
    	Ibw_h->bot[nb]   = I_blackC(Twb,narrBand[narrBand[nb].idx].wvc);
# else
    	Ibw_h->west[nb]  = I_blackC(Tww,narrBand[nb].wvc);
    	Ibw_h->east[nb]  = I_blackC(Twe,narrBand[nb].wvc);
    	Ibw_h->north[nb] = I_blackC(Twn,narrBand[nb].wvc);
    	Ibw_h->south[nb] = I_blackC(Tws,narrBand[nb].wvc);
    	Ibw_h->top[nb]   = I_blackC(Twt,narrBand[nb].wvc);
    	Ibw_h->bot[nb]   = I_blackC(Twb,narrBand[nb].wvc);
#endif
    }

	myfloat *west,*east;
	myfloat *north,*south;
	myfloat *top,*bot;
	EmissSpec *temp_Ib;

	temp_Ib = (EmissSpec*)malloc(nB*sizeof(EmissSpec));

	hipMalloc((void**)&west , nB * sizeof(myfloat));
	hipMalloc((void**)&east , nB * sizeof(myfloat));
	hipMalloc((void**)&north, nB * sizeof(myfloat));
	hipMalloc((void**)&south, nB * sizeof(myfloat));
	hipMalloc((void**)&top  , nB * sizeof(myfloat));
	hipMalloc((void**)&bot  , nB * sizeof(myfloat));

	hipMemcpy(west , Ibw_h->west , nB * sizeof(myfloat), hipMemcpyHostToDevice);
	hipMemcpy(east , Ibw_h->east , nB * sizeof(myfloat), hipMemcpyHostToDevice);
	hipMemcpy(north, Ibw_h->north, nB * sizeof(myfloat), hipMemcpyHostToDevice);
	hipMemcpy(south, Ibw_h->south, nB * sizeof(myfloat), hipMemcpyHostToDevice);
	hipMemcpy(top  , Ibw_h->top  , nB * sizeof(myfloat), hipMemcpyHostToDevice);
	hipMemcpy(bot  , Ibw_h->bot  , nB * sizeof(myfloat), hipMemcpyHostToDevice);

	temp_Ib->west  = west;
	temp_Ib->east  = east;
	temp_Ib->north = north;
	temp_Ib->south = south;
	temp_Ib->top   = top;
	temp_Ib->bot   = bot;

	hipMemcpy(Ibw_d, temp_Ib, nB * sizeof(EmissSpec), hipMemcpyHostToDevice);
	cudaCheckErrors("hipMalloc Ibw error");

	Ibw_h->destroyVar();
	free(Ibw_h);
        free(temp_Ib);
}


void grid_copy(Gridn *gridCPU, Gridn *gridGPU)
{
    myfloat *xGPU[grid_num],*xuGPU[grid_num];
    myfloat *yGPU[grid_num],*yuGPU[grid_num];
    myfloat *zGPU[grid_num],*zuGPU[grid_num];
    Gridn *temp_G;
    temp_G = (Gridn*)malloc(grid_num*sizeof(Gridn));

    for (int grd=0; grd<grid_num; grd++)
    {
    	hipMalloc((void**)&xGPU[grd] , (gridCPU[grd].im+2) * sizeof(myfloat));
    	hipMalloc((void**)&xuGPU[grd], (gridCPU[grd].im+1) * sizeof(myfloat));
    	hipMalloc((void**)&yGPU[grd] , (gridCPU[grd].jm+2) * sizeof(myfloat));
    	hipMalloc((void**)&yuGPU[grd], (gridCPU[grd].jm+1) * sizeof(myfloat));
    	hipMalloc((void**)&zGPU[grd] , (gridCPU[grd].km+2) * sizeof(myfloat));
    	hipMalloc((void**)&zuGPU[grd], (gridCPU[grd].km+1) * sizeof(myfloat));

    	hipMemcpy(xGPU[grd] , gridCPU[grd].x , (gridCPU[grd].im+2) * sizeof(myfloat), hipMemcpyHostToDevice);
    	hipMemcpy(xuGPU[grd], gridCPU[grd].xu, (gridCPU[grd].im+1) * sizeof(myfloat), hipMemcpyHostToDevice);
    	hipMemcpy(yGPU[grd] , gridCPU[grd].y , (gridCPU[grd].jm+2) * sizeof(myfloat), hipMemcpyHostToDevice);
    	hipMemcpy(yuGPU[grd], gridCPU[grd].yu, (gridCPU[grd].jm+1) * sizeof(myfloat), hipMemcpyHostToDevice);
    	hipMemcpy(zGPU[grd] , gridCPU[grd].z , (gridCPU[grd].km+2) * sizeof(myfloat), hipMemcpyHostToDevice);
    	hipMemcpy(zuGPU[grd], gridCPU[grd].zu, (gridCPU[grd].km+1) * sizeof(myfloat), hipMemcpyHostToDevice);

    	temp_G[grd].x  = xGPU[grd];
    	temp_G[grd].xu = xuGPU[grd];
    	temp_G[grd].y  = yGPU[grd];
    	temp_G[grd].yu = yuGPU[grd];
    	temp_G[grd].z  = zGPU[grd];
    	temp_G[grd].zu = zuGPU[grd];
    	temp_G[grd].im = gridCPU[grd].im;
    	temp_G[grd].jm = gridCPU[grd].jm;
    	temp_G[grd].km = gridCPU[grd].km;
    	temp_G[grd].sm = gridCPU[grd].sm;

    }
	hipMemcpy(gridGPU, temp_G, grid_num * sizeof(Gridn), hipMemcpyHostToDevice);
	cudaCheckErrors("hipMalloc Grid error");
        free(temp_G);
}

void temp_fluid_copy(hipTextureObject_t *tex_tempf_d, Var_CPU *varCPU)
{
     hipArray *T_volumeArray;
     static hipTextureObject_t tex_temp;
     
     //First grid temperature
     const hipExtent extentT = make_hipExtent(varCPU[0].im+2, varCPU[0].jm+2, varCPU[0].km+2);
     hipChannelFormatDesc channelDescT = hipCreateChannelDesc<myfloat>();
     hipMalloc3DArray(&T_volumeArray, &channelDescT, extentT);
     cudaCheckErrors("hipMalloc3D error");
     
     // Copying host memory to 3D cuda_Array
     hipMemcpy3DParms copyParamsT = {0};
     copyParamsT.srcPtr   = make_hipPitchedPtr((void*)varCPU[0].T, extentT.width*sizeof(myfloat), extentT.width, extentT.height);
     copyParamsT.dstArray = T_volumeArray;
     copyParamsT.extent   = extentT;
     copyParamsT.kind     = hipMemcpyHostToDevice;
     
     hipMemcpy3D(&copyParamsT);
     cudaCheckErrors("hipMemcpy3D fail");
     
     // Binding 3D cuda_Array to texture array
     hipResourceDesc    texRes;
     memset(&texRes, 0, sizeof(hipResourceDesc));
     texRes.resType = hipResourceTypeArray;
     texRes.res.array.array  = T_volumeArray;
     hipTextureDesc     texDescr;
     memset(&texDescr, 0, sizeof(hipTextureDesc));
     texDescr.normalizedCoords = false;
     texDescr.filterMode = hipFilterModeLinear;
     texDescr.addressMode[0] = hipAddressModeClamp;   // clamp
     texDescr.addressMode[1] = hipAddressModeClamp;
     texDescr.addressMode[2] = hipAddressModeClamp;
     texDescr.readMode = hipReadModeElementType;
     hipCreateTextureObject(&tex_temp, &texRes, &texDescr, NULL);
     cudaCheckErrors("Bind fail");
     hipMemcpy(tex_tempf_d , &tex_temp , sizeof(hipTextureObject_t) ,hipMemcpyHostToDevice);
}

void narrowband_copy(NarrowBand *narrBand, myfloat *wvc_d, int *idx_d, hipTextureObject_t *tex_d, hipTextureObject_t *tex_prob_d,
					myfloat *Tnb_d, myfloat *Tnb)
{
    myfloat *wvc_h;
    wvc_h = (myfloat*)malloc(nB*sizeof(myfloat));
    int *idx_h;
    idx_h = (int*)malloc(nB*sizeof(int));
    for(int nb = 0; nb<nB; nb++)
    	idx_h[nb] = narrBand[nb].idx;

    for(int nb = 0; nb<nB; nb++ )
    {
#if srt == 1
    	wvc_h[nb] = narrBand[narrBand[nb].idx].wvc;
#else
    	wvc_h[nb] = narrBand[nb].wvc;
#endif
    }

    hipMemcpy(idx_d   , idx_h   , nB *      sizeof(int)     ,hipMemcpyHostToDevice);
    hipMemcpy(wvc_d   , wvc_h   , nB *      sizeof(myfloat) ,hipMemcpyHostToDevice);
    hipMemcpy(Tnb_d   , Tnb     , nT *      sizeof(myfloat) ,hipMemcpyHostToDevice);

    free(idx_h);
    free(wvc_h);

    myfloat kq[nQ][nT][nB];

    for(int g = 0; g<nQ; g++)
    {
    	for(int t = 0; t<nT; t++)
		{
    		for(int nb = 0; nb<nB; nb++)
    		{
#if srt == 1
    			kq[g][t][nb] = narrBand[narrBand[nb].idx].kq[t][g];
#else
    			kq[g][t][nb] = narrBand[nb].kq[t][g];
#endif
    		}
		}
    }

    hipArray *d_volumeArray = 0;
    static hipTextureObject_t tex1_d;

    hipArray *PG_volumeArray = 0;
	static hipTextureObject_t tex1_prob_d;

	const hipExtent extent = make_hipExtent(nB, nT, nQ);
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<myfloat>();
	hipMalloc3DArray(&d_volumeArray, &channelDesc, extent);
	cudaCheckErrors("hipMalloc3D error");

	// Copying host memory to 3D cuda_Array
	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr   = make_hipPitchedPtr((void*)kq, extent.width*sizeof(myfloat), extent.width, extent.height);
	copyParams.dstArray = d_volumeArray;
	copyParams.extent   = extent;
	copyParams.kind     = hipMemcpyHostToDevice;

	hipMemcpy3D(&copyParams);
	cudaCheckErrors("hipMemcpy3D fail");

	// Binding 3D cuda_Array to texture array
	hipResourceDesc    texRes;
	memset(&texRes, 0, sizeof(hipResourceDesc));
	texRes.resType = hipResourceTypeArray;
	texRes.res.array.array  = d_volumeArray;
	hipTextureDesc     texDescr;
	memset(&texDescr, 0, sizeof(hipTextureDesc));
	texDescr.normalizedCoords = false;
	texDescr.filterMode = hipFilterModeLinear;
	texDescr.addressMode[0] = hipAddressModeClamp;   // clamp
	texDescr.addressMode[1] = hipAddressModeClamp;
	texDescr.addressMode[2] = hipAddressModeClamp;
	texDescr.readMode = hipReadModeElementType;
    hipCreateTextureObject(&tex1_d, &texRes, &texDescr, NULL);
    cudaCheckErrors("Bind fail");

    hipMemcpy(tex_d , &tex1_d , sizeof(hipTextureObject_t) ,hipMemcpyHostToDevice);


	const hipExtent extentP = make_hipExtent(nQ, nB, nT);
	hipChannelFormatDesc channelDescP = hipCreateChannelDesc<myfloat>();
	hipMalloc3DArray(&PG_volumeArray, &channelDescP, extentP);
	cudaCheckErrors("hipMalloc3D error");

	// Copying host memory to 3D cuda_Array
	hipMemcpy3DParms copyParamsP = {0};
	copyParamsP.srcPtr   = make_hipPitchedPtr((void*)probg, extentP.width*sizeof(myfloat), extentP.width, extentP.height);
	copyParamsP.dstArray = PG_volumeArray;
	copyParamsP.extent   = extentP;
	copyParamsP.kind     = hipMemcpyHostToDevice;

	hipMemcpy3D(&copyParamsP);
	cudaCheckErrors("hipMemcpy3D fail");

	// Binding 3D cuda_Array to texture array
	hipResourceDesc    texResP;
	memset(&texResP, 0, sizeof(hipResourceDesc));
	texResP.resType = hipResourceTypeArray;
	texResP.res.array.array  = PG_volumeArray;
	hipTextureDesc     texDescrP;
	memset(&texDescrP, 0, sizeof(hipTextureDesc));
	texDescrP.normalizedCoords = false;
	texDescrP.filterMode = hipFilterModeLinear;
	texDescrP.addressMode[0] = hipAddressModeClamp;   // clamp
	texDescrP.addressMode[1] = hipAddressModeClamp;
	texDescrP.addressMode[2] = hipAddressModeClamp;
	texDescrP.readMode = hipReadModeElementType;
    hipCreateTextureObject(&tex1_prob_d, &texResP, &texDescrP, NULL);
    cudaCheckErrors("Bind fail");

    hipMemcpy(tex_prob_d , &tex1_prob_d , sizeof(hipTextureObject_t) ,hipMemcpyHostToDevice);

}

myfloat I_blackC( myfloat T, myfloat nu)
{
	// way of calculating C1 and C2 (nu is in cm^(-1) while the output is in W/m^2

    myfloat C1 = 3.741771790075259e-16;
    myfloat C2 = 0.014387741858429;

	return 1.0 / pi * C1 * pow3(nu*100) / (expf(C2*nu*100/T)-1);

}
